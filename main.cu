// modified from the sample code at:
//  https://docs.nvidia.com/deeplearning/sdk/nccl-developer-guide/index.html#examples
#include <stdio.h>
#include <nccl.h>
#include <string>
#include <stdexcept>
#include <string.h>
#include <stdlib.h>

#define THROW(fmt, ...)                                         \
    do {                                                        \
        std::string msg;                                        \
        char errMsg[2048];                                      \
        sprintf(errMsg, "Exception occured! file=%s line=%d: ", \
                __FILE__, __LINE__);                            \
        msg += errMsg;                                          \
        sprintf(errMsg, fmt, ##__VA_ARGS__);                    \
        msg += errMsg;                                          \
        throw std::runtime_error(msg);                          \
    } while(0)

#define ASSERT(check, fmt, ...)                  \
    do {                                         \
        if(!(check))  THROW(fmt, ##__VA_ARGS__); \
    } while(0)

#define CUDA_CHECK(call)                                \
    do {                                                \
        hipError_t status = call;                      \
        ASSERT(status == hipSuccess,                   \
               "FAIL: call='%s'. Reason:%s\n",          \
               #call, hipGetErrorString(status));      \
    } while(0)

#define NCCL_CHECK(cmd)                                   \
    do {                                                  \
        ncclResult_t status = cmd;                        \
        ASSERT(status == ncclSuccess,                     \
               "FAIL: nccl-call='%s'. Reason:%s\n",       \
               #call, ncclGetErrorString(r));             \
    } while(0)

bool isPo2(int in) {
    return (in > 1) && (in & in-1);
}

void printHelp() {
    printf("USAGE:\n");
    printf(" ./nccl [-h] [-n <nDevices>] [-s <buffSize>]\n");
}

int main(int argc, char** argv) {
    int nDevices = 2;
    int size = 32*1024*1024;
    for(int i=1;i<argc;++i) {
        if(!strcmp("-h", argv[i])) {
            printHelp();
            return 0;
        } else if(!strcmp("-n", argv[i])) {
            ASSERT(i < argc, "'-n' requires an argument!");
            ++i;
            nDevices = atoi(argv[i]);
            ASSERT(isPo2(nDevices), "Num devices must be PO2 and more than 1!");
        } else if(!strcmp("-s", argv[i])) {
            ASSERT(i < argc, "'-s' requires an argument!");
            ++i;
            size = atoi(argv[i]);
        } else {
            ASSERT(false, "Incorrect argument '%s'!", argv[i]);
        }
    }
    ncclComm_t* comms = new ncclComm_t[nDevices];
    int* devs = new int[nDevices];
    for(int i=0;i<nDevices;++i) {
        devs[i] = i;
    }
    hipStream_t* streams = new hipStream_t[nDevices];
    int** sendbuff = new int*[nDevices];
    int** recvbuff = new int*[nDevices];
    hipEvent_t* events = new hipEvent_t[2*nDevices];
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(sendbuff+i, size*sizeof(int)));
        CUDA_CHECK(hipMalloc(recvbuff + i, size*sizeof(int)));
        CUDA_CHECK(hipMemset(sendbuff[i], 1, size*sizeof(int)));
        CUDA_CHECK(hipMemset(recvbuff[i], 0, size*sizeof(int)));
        CUDA_CHECK(hipStreamCreate(streams+i));
        CUDA_CHECK(hipEventCreate(events+2*i));
        CUDA_CHECK(hipEventCreate(events+2*i+1));
    }
    NCCL_CHECK(ncclCommInitAll(comms, nDevices, devs));
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipEventRecord(events[2*i], streams[i]));
    }
    NCCL_CHECK(ncclGroupStart());
    for(int i=0;i<nDevices;++i) {
        NCCL_CHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i],
                                 size, ncclInt, ncclSum, comms[i], streams[i]));
    }
    NCCL_CHECK(ncclGroupEnd());
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipEventRecord(events[2*i+1], streams[i]));
    }
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipEventSynchronize(events[2*i+1]));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        float et;
        CUDA_CHECK(hipEventElapsedTime(&et, events[2*i], events[2*i+1]));
        printf("Device=%d,nDevices=%d,size=%d,time=%fms\n", i, nDevices, size, et);
    }
    for(int i=0;i<nDevices;++i) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipFree(recvbuff[i]));
        CUDA_CHECK(hipFree(sendbuff[i]));
        CUDA_CHECK(hipEventDestroy(events[2*i]));
        CUDA_CHECK(hipEventDestroy(events[2*i+1]));
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    for(int i=0;i<nDevices;++i) {
        NCCL_CHECK(ncclCommDestroy(comms[i]));
    }
    delete [] events;
    delete [] recvbuff;
    delete [] sendbuff;
    delete [] streams;
    delete [] devs;
    delete [] comms;
    printf("Success\n");
    return 0;
}
